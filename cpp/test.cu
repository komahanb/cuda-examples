#include "hip/hip_runtime.h"
#include <iostream>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include "index.cuh"

/**
   Function to add integer arrays
*/
__global__ void add_array(int *a, int *b, int size){
  for (int i = 0; i < size; i++){
    a[i] += b[i];
  }
}

/**
   Function to set thread index for intialization of arrays
*/
__global__ void set_arrays(int *a, int *b, int size){  
  int i = getGlobalIdx_1D_1D(); // stored in register memory
  if (i >= 0 && i < size){
    a[i] = i;
    b[i] = i;
  }
}

int main(){
  // allocated on CPU memory space (Host)
  // int a[]={12}, b[]={1};

  static const int N = 10000;
  int *a = new int[N];
  int *b = new int[N];

  // allocated on GPU memory space (Device)
  int *d_a, *d_b;  
  hipMalloc(&d_a, N*sizeof(int));
  hipMalloc(&d_b, N*sizeof(int));

  // copy data to gpu for computations
  // hipMemcpy(d_a, &a, N*sizeof(int), hipMemcpyHostToDevice);
  // hipMemcpy(d_b, &b, N*sizeof(int), hipMemcpyHostToDevice);

  // perform computations using kernel written for GPU
  // add_array<<<1,1>>>(d_a, d_b, 1);
  dim3 block_dim(10,1,1);
  dim3 grid_dim(N/10,1,1);
  set_arrays<<<block_dim, grid_dim>>>(d_a, d_b, N);
  
  // copy result stored in d_a back to cpu for printing
  hipMemcpy(a, d_a, N*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(b, d_b, N*sizeof(int), hipMemcpyDeviceToHost);
  
  // print the result
  for (int i = N-10; i < N; i++){
    printf("result[%d]=%d\n", i, a[i]);
  }
  
  // free up device memory
  hipFree(d_a);
  hipFree(d_b);

  delete [] a;
  delete [] b;
    
  return 0;
}
